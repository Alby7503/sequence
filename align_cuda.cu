#include "hip/hip_runtime.h"
/*
 * Exact genetic sequence alignment
 * (Using brute force)
 *
 * CUDA version
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2023/2024
 *
 * v1.3
 *
 * (c) 2024, Arturo Gonzalez-Escribano
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <limits.h>
#include <sys/time.h>

/* Headers for the CUDA assignment versions */
#include <hip/hip_runtime.h> // Libreria principale per la programmazione CUDA

/* Macro per controllare gli errori delle chiamate a funzioni CUDA.
 * Esegue la funzione e, se l'esito non è 'hipSuccess', stampa un messaggio di errore
 * con la linea del codice e la descrizione dell'errore. */
#define CUDA_CHECK_FUNCTION(call)                                                                 \
    {                                                                                             \
        hipError_t check = call;                                                                 \
        if (check != hipSuccess)                                                                 \
            fprintf(stderr, "CUDA Error in line: %d, %s\n", __LINE__, hipGetErrorString(check)); \
    }
/* Macro per controllare gli errori dopo il lancio di un kernel CUDA.
 * I kernel vengono eseguiti in modo asincrono, quindi 'hipGetLastError'
 * è necessario per verificare se il kernel lanciato in precedenza ha generato un errore. */
#define CUDA_CHECK_KERNEL()                                                                              \
    {                                                                                                    \
        hipError_t check = hipGetLastError();                                                          \
        if (check != hipSuccess)                                                                        \
            fprintf(stderr, "CUDA Kernel Error in line: %d, %s\n", __LINE__, hipGetErrorString(check)); \
    }

/* Valore speciale per indicare che un pattern non è stato trovato nella sequenza. */
#define NOT_FOUND -1

// #define DEBUG // Se decommentata, abilita la stampa di informazioni di debug

/* Valore massimo per il calcolo dei checksum, per evitare overflow e mantenere i valori in un range gestibile. */
#define CHECKSUM_MAX 65535

/*
 * Utils: Funzione per ottenere il tempo di orologio (wall time).
 * Utile per misurare le performance del codice.
 */
double cp_Wtime()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + 1.0e-6 * tv.tv_usec;
}

/*
 * Utils: Inclusione del generatore di numeri casuali.
 */
#include "rng.c"

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 * DO NOT USE OpenMP IN YOUR CODE
 *
 */

__global__ void generate_rng_sequence_kernel(rng_t *d_random, float prob_G, float prob_C, float prob_A, char *d_seq, unsigned long length)
{
    // Calcola l'ID univoco del thread corrente.
    unsigned long ind = blockIdx.x * blockDim.x + threadIdx.x;

    // Ogni thread genera un carattere della sequenza.
    if (ind < length)
    {
        rng_t local_rng = *d_random; // Copia il generatore di numeri casuali dalla memoria della GPU.
        rng_skip(&local_rng, ind); // Salta il generatore di numeri casuali per ottenere un carattere unico.
        double prob = rng_next(&local_rng);
        //__syncthreads(); // Sincronizza i thread per garantire che il generatore di numeri casuali sia aggiornato.
        if (prob < prob_G)
            d_seq[ind] = 'G';
        else if (prob < prob_C)
            d_seq[ind] = 'C';
        else if (prob < prob_A)
            d_seq[ind] = 'A';
        else
            d_seq[ind] = 'T';
    }
}

/*
 * KERNEL CUDA: Cerca la prima corrispondenza per ogni pattern in parallelo.
 * '__global__' indica che questa funzione viene eseguita sulla GPU e può essere chiamata dalla CPU.
 * Parametri: puntatori a dati che risiedono nella memoria della GPU.
 */
__global__ void find_patterns_kernel(const char *d_sequence, unsigned long seq_length,
                                     char **d_pattern, const unsigned long *d_pat_length,
                                     int pat_number, unsigned long *d_pat_found)
{
    // Calcola l'ID univoco del thread corrente. Questo ID corrisponde all'indice del pattern di cui si occuperà.
    int pat_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // "Guardia": se l'ID del thread è maggiore del numero di pattern, il thread non fa nulla ed esce.
    // Questo è necessario perché il numero totale di thread lanciati potrebbe essere maggiore del numero di pattern.
    if (pat_idx >= pat_number)
    {
        return;
    }

    // Ogni thread ottiene i dati specifici per il suo pattern: la lunghezza e il puntatore al contenuto del pattern.
    unsigned long my_pat_length = d_pat_length[pat_idx];
    char *my_pattern = d_pattern[pat_idx];

    // Loop principale di ricerca (brute-force): scorre tutte le possibili posizioni di partenza nella sequenza.
    for (unsigned long start = 0; start <= seq_length - my_pat_length; start++)
    {
        unsigned long lind;
        // Loop interno: confronta carattere per carattere il pattern con la sottosequenza corrente.
        for (lind = 0; lind < my_pat_length; lind++)
        {
            // Se un carattere non corrisponde, interrompe il confronto per questa posizione di partenza.
            if (d_sequence[start + lind] != my_pattern[lind])
            {
                break;
            }
        }
        // Se il loop interno è terminato perché tutti i caratteri corrispondevano (lind == my_pat_length)...
        if (lind == my_pat_length)
        {
            // ...il pattern è stato trovato. Salva la posizione di inizio nell'array dei risultati.
            d_pat_found[pat_idx] = start;
            // Questo thread ha finito il suo compito (trovare la *prima* corrispondenza), quindi termina.
            return;
        }
    }
}

/*
 * KERNEL CUDA: Aggiorna l'array seq_matches in modo atomico.
 * Viene eseguito dopo find_patterns_kernel.
 */
__global__ void increment_matches_kernel(const unsigned long *d_pat_found, const unsigned long *d_pat_length,
                                         int pat_number, int *d_seq_matches)
{
    // Calcola l'ID univoco del thread, che corrisponde all'indice del pattern.
    int pat_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pat_idx >= pat_number)
    {
        return;
    }

    // Il thread procede solo se il suo pattern è stato effettivamente trovato nel kernel precedente.
    if (d_pat_found[pat_idx] != (unsigned long)NOT_FOUND)
    {
        // Ottiene la posizione e la lunghezza del pattern trovato.
        unsigned long start_pos = d_pat_found[pat_idx];
        unsigned long length = d_pat_length[pat_idx];

        // Itera su ogni posizione della sequenza coperta da questo pattern.
        for (unsigned long ind = 0; ind < length; ind++)
        {
            // 'addr' è il puntatore alla cella di memoria in 'd_seq_matches' da aggiornare.
            int *addr = &d_seq_matches[start_pos + ind];

            // Inizia il "CAS loop" per un aggiornamento atomico sicuro.
            // Legge il valore corrente. Questa lettura non è atomica, ma serve solo come primo tentativo.
            int old_val = *addr;

            // Loop che continua finché l'aggiornamento atomico non va a buon fine.
            while (true)
            {
                // Determina il nuovo valore desiderato secondo la logica sequenziale.
                int new_val;
                if (old_val == NOT_FOUND) // Se è il primo pattern a coprire questa posizione
                {
                    new_val = 1;
                }
                else // Se altri pattern hanno già coperto questa posizione
                {
                    new_val = old_val + 1;
                }

                // Tenta di eseguire un'operazione atomica di Compare-And-Swap (CAS).
                // Sostituisce il valore in 'addr' con 'new_val' SOLO SE il valore attuale è ancora 'old_val'.
                // Restituisce il valore che c'era in memoria PRIMA dell'operazione.
                int current_val_in_mem = atomicCAS(addr, old_val, new_val);

                // Se il valore prima del CAS era quello che ci aspettavamo, l'operazione è riuscita.
                if (current_val_in_mem == old_val)
                {
                    break; // Successo! L'aggiornamento è completo, esci dal loop.
                }

                // Se il CAS è fallito, significa che un altro thread ha modificato il valore nel frattempo.
                // Aggiorna 'old_val' con il valore più recente e riprova il loop.
                old_val = current_val_in_mem;
            }
        }
    }
}

/*
 * KERNEL CUDA: Inizializza gli array dei risultati sulla GPU.
 * È più efficiente farlo sulla GPU che sulla CPU e poi trasferire.
 */
__global__ void initialize_arrays_kernel(unsigned long *d_pat_found, int *d_seq_matches,
                                         int pat_number, unsigned long seq_length)
{
    // Calcola l'ID univoco del thread.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Inizializza l'array d_pat_found.
    if (idx < pat_number)
    {
        d_pat_found[idx] = (unsigned long)NOT_FOUND;
    }

    // Inizializza l'array d_seq_matches usando un "grid-stride loop".
    // Questo pattern garantisce che ogni elemento dell'array venga inizializzato
    // anche se il numero di elementi è maggiore del numero di thread.
    for (unsigned long i = idx; i < seq_length; i += gridDim.x * blockDim.x)
    {
        d_seq_matches[i] = NOT_FOUND;
    }
}

/*
 * Funzione CPU (ora vuota): L'aggiornamento dei match.
 * Nella versione CUDA, la sua logica è stata spostata nel kernel 'increment_matches_kernel'.
 * Viene lasciata vuota per mantenere la compatibilità con la struttura del template.
 */
void increment_matches(int pat, unsigned long *pat_found, const unsigned long *pat_length, int *seq_matches)
{
    unsigned long ind;
    for (ind = 0; ind < pat_length[pat]; ind++)
    {
        if (seq_matches[pat_found[pat] + ind] == NOT_FOUND)
            seq_matches[pat_found[pat] + ind] = 0;
        else
            seq_matches[pat_found[pat] + ind]++;
    }
}

/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

/*
 * Function: Allocate new patttern
 */
char *pattern_allocate(rng_t *random, unsigned long pat_rng_length_mean, unsigned long pat_rng_length_dev, unsigned long seq_length, unsigned long *new_length)
{

    /* Random length */
    unsigned long length = (unsigned long)rng_next_normal(random, (double)pat_rng_length_mean, (double)pat_rng_length_dev);
    if (length > seq_length)
        length = seq_length;
    if (length <= 0)
        length = 1;

    /* Allocate pattern */
    char *pattern = (char *)malloc(sizeof(char) * length);
    if (pattern == NULL)
    {
        fprintf(stderr, "\n-- Error allocating a pattern of size: %lu\n", length);
        exit(EXIT_FAILURE);
    }

    /* Return results */
    *new_length = length;
    return pattern;
}

/*
 * Function: Fill random sequence or pattern
 */
void generate_rng_sequence(rng_t *random, float prob_G, float prob_C, float prob_A, char *seq, unsigned long length)
{
    unsigned long ind;
    for (ind = 0; ind < length; ind++)
    {
        double prob = rng_next(random);
        if (prob < prob_G)
            seq[ind] = 'G';
        else if (prob < prob_C)
            seq[ind] = 'C';
        else if (prob < prob_A)
            seq[ind] = 'A';
        else
            seq[ind] = 'T';
    }
}

/*
 * Function: Copy a sample of the sequence
 */
void copy_sample_sequence(rng_t *random, char *sequence, unsigned long seq_length, unsigned long pat_samp_loc_mean, unsigned long pat_samp_loc_dev, char *pattern, unsigned long length)
{
    /* Choose location */
    unsigned long location = (unsigned long)rng_next_normal(random, (double)pat_samp_loc_mean, (double)pat_samp_loc_dev);
    if (location > seq_length - length)
        location = seq_length - length;
    if (location <= 0)
        location = 0;

    /* Copy sample */
    unsigned long ind;
    for (ind = 0; ind < length; ind++)
        pattern[ind] = sequence[ind + location];
}

/*
 * Function: Regenerate a sample of the sequence
 */
void generate_sample_sequence(rng_t *random, rng_t random_seq, float prob_G, float prob_C, float prob_A, unsigned long seq_length, unsigned long pat_samp_loc_mean, unsigned long pat_samp_loc_dev, char *pattern, unsigned long length)
{
    /* Choose location */
    unsigned long location = (unsigned long)rng_next_normal(random, (double)pat_samp_loc_mean, (double)pat_samp_loc_dev);
    if (location > seq_length - length)
        location = seq_length - length;
    if (location <= 0)
        location = 0;

    /* Regenerate sample */
    rng_t local_random = random_seq;
    rng_skip(&local_random, location);
    generate_rng_sequence(&local_random, prob_G, prob_C, prob_A, pattern, length);
}

/*
 * Function: Print usage line in stderr
 */
void show_usage(char *program_name)
{
    fprintf(stderr, "Usage: %s ", program_name);
    fprintf(stderr, "<seq_length> <prob_G> <prob_C> <prob_A> <pat_rng_num> <pat_rng_length_mean> <pat_rng_length_dev> <pat_samples_num> <pat_samp_length_mean> <pat_samp_length_dev> <pat_samp_loc_mean> <pat_samp_loc_dev> <pat_samp_mix:B[efore]|A[fter]|M[ixed]> <long_seed>\n");
    fprintf(stderr, "\n");
}

/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[])
{
    /* 0. Disabilita il buffering per stdout e stderr per avere output immediato. */
    setbuf(stdout, NULL);
    setbuf(stderr, NULL);

    /* 1. Lettura e validazione degli argomenti dalla riga di comando. */
    if (argc < 15)
    {
        fprintf(stderr, "\n-- Error: Not enough arguments when reading configuration from the command line\n\n");
        show_usage(argv[0]);
        exit(EXIT_FAILURE);
    }

    /* 1.2. Lettura dei valori degli argomenti. */
    unsigned long seq_length = atol(argv[1]);
    float prob_G = atof(argv[2]);
    float prob_C = atof(argv[3]);
    float prob_A = atof(argv[4]);
    if (prob_G + prob_C + prob_A > 1)
    {
        fprintf(stderr, "\n-- Error: The sum of G,C,A,T nucleotid probabilities cannot be higher than 1\n\n");
        show_usage(argv[0]);
        exit(EXIT_FAILURE);
    }
    prob_C += prob_G;
    prob_A += prob_C;

    int pat_rng_num = atoi(argv[5]);
    unsigned long pat_rng_length_mean = atol(argv[6]);
    unsigned long pat_rng_length_dev = atol(argv[7]);

    int pat_samp_num = atoi(argv[8]);
    unsigned long pat_samp_length_mean = atol(argv[9]);
    unsigned long pat_samp_length_dev = atol(argv[10]);
    unsigned long pat_samp_loc_mean = atol(argv[11]);
    unsigned long pat_samp_loc_dev = atol(argv[12]);

    char pat_samp_mix = argv[13][0];
    if (pat_samp_mix != 'B' && pat_samp_mix != 'A' && pat_samp_mix != 'M')
    {
        fprintf(stderr, "\n-- Error: Incorrect first character of pat_samp_mix: %c\n\n", pat_samp_mix);
        show_usage(argv[0]);
        exit(EXIT_FAILURE);
    }

    unsigned long seed = atol(argv[14]);

#ifdef DEBUG
    /* DEBUG: Stampa degli argomenti letti, se la modalità DEBUG è attiva. */
    printf("\nArguments: seq_length=%lu\n", seq_length);
    printf("Arguments: Accumulated probabilitiy G=%f, C=%f, A=%f, T=1\n", prob_G, prob_C, prob_A);
    printf("Arguments: Random patterns number=%d, length_mean=%lu, length_dev=%lu\n", pat_rng_num, pat_rng_length_mean, pat_rng_length_dev);
    printf("Arguments: Sample patterns number=%d, length_mean=%lu, length_dev=%lu, loc_mean=%lu, loc_dev=%lu\n", pat_samp_num, pat_samp_length_mean, pat_samp_length_dev, pat_samp_loc_mean, pat_samp_loc_dev);
    printf("Arguments: Type of mix: %c, Random seed: %lu\n", pat_samp_mix, seed);
    printf("\n");
#endif // DEBUG

    /* Seleziona il dispositivo GPU da usare (in questo caso il dispositivo 0). */
    CUDA_CHECK_FUNCTION(hipSetDevice(0));
    CUDA_CHECK_FUNCTION(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

    /* 2. Inizializzazione delle strutture dati sulla CPU (Host). */
    rng_t random = rng_new(seed);
    // Salta il generatore di numeri casuali per la lunghezza della sequenza, per mantenere
    // lo stato del generatore identico a quello della versione sequenziale.
    rng_skip(&random, seq_length);

    /* 2.2. Alloca e genera i pattern sulla CPU. */
    int pat_number = pat_rng_num + pat_samp_num;
    unsigned long *pat_length = (unsigned long *)malloc(sizeof(unsigned long) * pat_number);
    char **pattern = (char **)malloc(sizeof(char *) * pat_number);
    if (pattern == NULL || pat_length == NULL)
    {
        fprintf(stderr, "\n-- Error allocating the basic patterns structures for size: %d\n", pat_number);
        exit(EXIT_FAILURE);
    }

    /* 2.2.2 Allocazione e inizializzazione di una struttura ausiliaria per il tipo di pattern. */
    int ind;
#define PAT_TYPE_NONE 0
#define PAT_TYPE_RNG 1
#define PAT_TYPE_SAMP 2
    char *pat_type = (char *)malloc(sizeof(char) * pat_number);
    if (pat_type == NULL)
    {
        fprintf(stderr, "\n-- Error allocating ancillary structure for pattern of size: %d\n", pat_number);
        exit(EXIT_FAILURE);
    }
    for (ind = 0; ind < pat_number; ind++)
        pat_type[ind] = PAT_TYPE_NONE;

    /* 2.2.3 Determina l'ordine dei pattern (casuali o campioni) in base al parametro 'pat_samp_mix'. */
    switch (pat_samp_mix)
    {
    case 'A':
        for (ind = 0; ind < pat_rng_num; ind++)
            pat_type[ind] = PAT_TYPE_RNG;
        for (; ind < pat_number; ind++)
            pat_type[ind] = PAT_TYPE_SAMP;
        break;
    case 'B':
        for (ind = 0; ind < pat_samp_num; ind++)
            pat_type[ind] = PAT_TYPE_SAMP;
        for (; ind < pat_number; ind++)
            pat_type[ind] = PAT_TYPE_RNG;
        break;
    default:
        if (pat_rng_num == 0)
        {
            for (ind = 0; ind < pat_number; ind++)
                pat_type[ind] = PAT_TYPE_SAMP;
        }
        else if (pat_samp_num == 0)
        {
            for (ind = 0; ind < pat_number; ind++)
                pat_type[ind] = PAT_TYPE_RNG;
        }
        else if (pat_rng_num < pat_samp_num)
        {
            int interval = pat_number / pat_rng_num;
            for (ind = 0; ind < pat_number; ind++)
                if ((ind + 1) % interval == 0)
                    pat_type[ind] = PAT_TYPE_RNG;
                else
                    pat_type[ind] = PAT_TYPE_SAMP;
        }
        else
        {
            int interval = pat_number / pat_samp_num;
            for (ind = 0; ind < pat_number; ind++)
                if ((ind + 1) % interval == 0)
                    pat_type[ind] = PAT_TYPE_SAMP;
                else
                    pat_type[ind] = PAT_TYPE_RNG;
        }
    }

    /* 2.2.4 Genera i pattern sulla CPU in base al tipo determinato in precedenza. */
    for (ind = 0; ind < pat_number; ind++)
    {
        if (pat_type[ind] == PAT_TYPE_RNG)
        {
            pattern[ind] = pattern_allocate(&random, pat_rng_length_mean, pat_rng_length_dev, seq_length, &pat_length[ind]);
            generate_rng_sequence(&random, prob_G, prob_C, prob_A, pattern[ind], pat_length[ind]);
        }
        else if (pat_type[ind] == PAT_TYPE_SAMP)
        {
            pattern[ind] = pattern_allocate(&random, pat_samp_length_mean, pat_samp_length_dev, seq_length, &pat_length[ind]);
// Il template forza la rigenerazione dei campioni, quindi questa opzione è sempre attiva.
#define REGENERATE_SAMPLE_PATTERNS
#ifdef REGENERATE_SAMPLE_PATTERNS
            rng_t random_seq_orig = rng_new(seed);
            generate_sample_sequence(&random, random_seq_orig, prob_G, prob_C, prob_A, seq_length, pat_samp_loc_mean, pat_samp_loc_dev, pattern[ind], pat_length[ind]);
#else
            // Questa parte non viene usata ma è mantenuta per completezza.
            // copy_sample_sequence( &random, sequence, seq_length, pat_samp_loc_mean, pat_samp_loc_dev, pattern[ind], pat_length[ind] );
#endif
        }
        else
        {
            fprintf(stderr, "\n-- Error internal: Paranoic check! A pattern without type at position %d\n", ind);
            exit(EXIT_FAILURE);
        }
    }
    free(pat_type); // Libera la memoria della struttura ausiliaria.

    printf("\n--- CUDA DEBUG ---\n");
    if (pat_number > 0)
    {
        printf("CUDA_DEBUG: Pattern 0[0] = %c\n", pattern[0][0]);
    }
    if (pat_number > 1)
    {
        printf("CUDA_DEBUG: Pattern 1[0] = %c\n", pattern[1][0]);
    }
    printf("--- END PATTERN DEBUG ---\n");

    /* Azzera gli argomenti per non usarli più (buona pratica richiesta dal problema). */
    argc = 0;
    argv = NULL;
    pat_rng_num = 0;
    pat_rng_length_mean = 0;
    pat_rng_length_dev = 0;
    pat_samp_num = 0;
    pat_samp_length_mean = 0;
    pat_samp_length_dev = 0;
    pat_samp_loc_mean = 0;
    pat_samp_loc_dev = 0;
    pat_samp_mix = '0';

    /* 2.3. Alloca memoria sulla CPU per gli array che conterranno i risultati. */
    unsigned long *pat_found;
    pat_found = (unsigned long *)malloc(sizeof(unsigned long) * pat_number);
    if (pat_found == NULL)
    {
        fprintf(stderr, "\n-- Error allocating aux pattern structure for size: %d\n", pat_number);
        exit(EXIT_FAILURE);
    }
    int *seq_matches;
    seq_matches = (int *)malloc(sizeof(int) * seq_length);
    if (seq_matches == NULL)
    {
        fprintf(stderr, "\n-- Error allocating aux sequence structures for size: %lu\n", seq_length);
        exit(EXIT_FAILURE);
    }

    /* 3. Sincronizza la GPU e fa partire il timer per misurare il tempo di calcolo. */
    CUDA_CHECK_FUNCTION(hipDeviceSynchronize());
    double ttotal = cp_Wtime();

    /*
     *
     * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
     * DO NOT USE OpenMP IN YOUR CODE
     *
     */
    /* 2.1. Ora genera la sequenza principale sulla CPU (Host). */
    char *sequence_h = (char *)malloc(sizeof(char) * seq_length);
    if (sequence_h == NULL)
    {
        fprintf(stderr, "\n-- Error allocating the sequence for size: %lu\n", seq_length);
        exit(EXIT_FAILURE);
    }

    //random = rng_new(seed);
    //generate_rng_sequence(&random, prob_G, prob_C, prob_A, sequence, seq_length);

    /* Lancia il kernel CUDA per generare la sequenza sulla GPU. */
    rng_t random_seq = rng_new(seed);
    rng_t *d_random;
    char *d_sequence;
    CUDA_CHECK_FUNCTION(hipMalloc(&d_random, sizeof(rng_t)));
    CUDA_CHECK_FUNCTION(hipMemcpy(d_random, &random_seq, sizeof(rng_t), hipMemcpyHostToDevice));
    CUDA_CHECK_FUNCTION(hipMalloc(&d_sequence, sizeof(char) * seq_length)); // Alloca memoria sulla GPU per la sequenza.
    unsigned long threads_per_block = 256;
    unsigned long grid_size = (seq_length + threads_per_block - 1) / threads_per_block;
    generate_rng_sequence_kernel<<<grid_size, threads_per_block>>>(d_random, prob_G, prob_C, prob_A, d_sequence, seq_length);
    CUDA_CHECK_KERNEL(); // Controlla errori dopo il lancio
    CUDA_CHECK_FUNCTION(hipMemcpy(&random_seq, d_random, sizeof(rng_t), hipMemcpyDeviceToHost));
    CUDA_CHECK_FUNCTION(hipFree(d_random)); // Libera la memoria allocata per il generatore di numeri casuali sulla GPU.
    CUDA_CHECK_FUNCTION(hipMemcpy(sequence_h, d_sequence, sizeof(char) * seq_length, hipMemcpyDeviceToHost)); // Copia la sequenza generata dalla GPU alla CPU.
    //sequence_h[seq_length - 1] = '\0';
    //printf("Generated sequence: %s\n", sequence_h); // Stampa la sequenza generata.

#ifdef DEBUG
    /* DEBUG: Stampa la sequenza e i pattern generati, se in modalità DEBUG. */
    printf("-----------------\n");
    printf("Sequence: ");
    for (unsigned long lind = 0; lind < seq_length; lind++)
        printf("%c", sequence[lind]);
    printf("\n-----------------\n");
    printf("Patterns: %d\n", pat_number);
    int debug_pat;
    for (debug_pat = 0; debug_pat < pat_number; debug_pat++)
    {
        printf("Pat[%d]: ", debug_pat);
        for (unsigned long lind = 0; lind < pat_length[debug_pat]; lind++)
            printf("%c", pattern[debug_pat][lind]);
        printf("\n");
    }
    printf("-----------------\n\n");
#endif // DEBUG

    /* Dichiarazione dei puntatori per la memoria della GPU (Device). */
    //char *d_sequence;
    unsigned long *d_pat_length;
    char **d_pattern;
    unsigned long *d_pat_found;
    int *d_seq_matches;

    /* Allocazione della memoria sulla GPU per tutti gli array necessari. */
    //CUDA_CHECK_FUNCTION(hipMalloc(&d_sequence, sizeof(char) * seq_length));
    CUDA_CHECK_FUNCTION(hipMalloc(&d_pat_length, sizeof(unsigned long) * pat_number));
    CUDA_CHECK_FUNCTION(hipMalloc(&d_pattern, sizeof(char *) * pat_number));
    CUDA_CHECK_FUNCTION(hipMalloc(&d_pat_found, sizeof(unsigned long) * pat_number));
    CUDA_CHECK_FUNCTION(hipMalloc(&d_seq_matches, sizeof(int) * seq_length));

    /* Trasferimento dei dati di input dalla CPU (Host) alla GPU (Device). */
    //CUDA_CHECK_FUNCTION(hipMemcpy(d_sequence, sequence, sizeof(char) * seq_length, hipMemcpyHostToDevice));
    CUDA_CHECK_FUNCTION(hipMemcpy(d_pat_length, pat_length, sizeof(unsigned long) * pat_number, hipMemcpyHostToDevice));

    /* Trasferimento dell'array di pattern (jagged array), che richiede una procedura a due passaggi. */
    // 1. Alloca un array temporaneo sulla CPU per contenere i puntatori della GPU.
    char **d_pattern_in_host = (char **)malloc(sizeof(char *) * pat_number);
    if (d_pattern_in_host == NULL)
    {
        fprintf(stderr, "\n-- Error allocating host-side device pointers array for size: %d\n", pat_number);
        exit(EXIT_FAILURE);
    }
    // 2. Per ogni pattern...
    for (int i = 0; i < pat_number; i++)
    {
        // ...alloca memoria sulla GPU per il singolo pattern...
        CUDA_CHECK_FUNCTION(hipMalloc(&(d_pattern_in_host[i]), sizeof(char) * pat_length[i]));
        // ...e copia il suo contenuto dalla CPU alla GPU.
        CUDA_CHECK_FUNCTION(hipMemcpy(d_pattern_in_host[i], pattern[i], sizeof(char) * pat_length[i], hipMemcpyHostToDevice));
    }
    // 3. Infine, copia l'array di puntatori (che ora puntano a locazioni di memoria GPU) dalla CPU alla GPU.
    CUDA_CHECK_FUNCTION(hipMemcpy(d_pattern, d_pattern_in_host, sizeof(char *) * pat_number, hipMemcpyHostToDevice));

    /* Configurazione per il lancio dei kernel CUDA: numero di thread per blocco e numero di blocchi nella griglia. */
    //int threads_per_block = 256;
    int grid_size_pat = (pat_number + threads_per_block - 1) / threads_per_block;
    int grid_size_seq = (seq_length + threads_per_block - 1) / threads_per_block;

    /* ESECUZIONE DEI KERNEL */

    // 1. Lancia il kernel per inizializzare gli array dei risultati sulla GPU.
    initialize_arrays_kernel<<<grid_size_seq, threads_per_block>>>(d_pat_found, d_seq_matches, pat_number, seq_length);
    CUDA_CHECK_KERNEL(); // Controlla errori dopo il lancio
    // REMOVE
    CUDA_CHECK_FUNCTION(hipDeviceSynchronize()); // Sincronizza la GPU per assicurarsi che l'inizializzazione sia completata.
    printf("initialize_arrays_kernel\n");

    // 2. Lancia il kernel per trovare i pattern.
    find_patterns_kernel<<<grid_size_pat, threads_per_block>>>(d_sequence, seq_length, d_pattern, d_pat_length, pat_number, d_pat_found);
    CUDA_CHECK_KERNEL();
    // REMOVE
    CUDA_CHECK_FUNCTION(hipDeviceSynchronize()); // Sincronizza la GPU per assicurarsi che l'inizializzazione sia completata.
    printf("find_patterns_kernel\n");

    // 3. Lancia il kernel per aggiornare i contatori dei match.
    increment_matches_kernel<<<grid_size_pat, threads_per_block>>>(d_pat_found, d_pat_length, pat_number, d_seq_matches);
    CUDA_CHECK_KERNEL();
    // REMOVE
    CUDA_CHECK_FUNCTION(hipDeviceSynchronize()); // Sincronizza la GPU per assicurarsi che l'inizializzazione sia completata.
    printf("increment_matches_kernel\n");

    /* Trasferimento dei risultati dalla GPU (Device) alla CPU (Host) per il calcolo finale e la stampa. */
    CUDA_CHECK_FUNCTION(hipMemcpy(pat_found, d_pat_found, sizeof(unsigned long) * pat_number, hipMemcpyDeviceToHost));
    CUDA_CHECK_FUNCTION(hipMemcpy(seq_matches, d_seq_matches, sizeof(int) * seq_length, hipMemcpyDeviceToHost));

    /* Calcoli finali sulla CPU basati sui risultati ottenuti dalla GPU. */
    int pat_matches = 0;
    unsigned long checksum_matches = 0;
    unsigned long checksum_found = 0;

    // Calcola il numero totale di pattern trovati e il checksum delle posizioni.
    for (ind = 0; ind < pat_number; ind++)
    {
        if (pat_found[ind] != (unsigned long)NOT_FOUND)
        {
            pat_matches++;
            checksum_found = (checksum_found + pat_found[ind]) % CHECKSUM_MAX;
        }
    }

    // Calcola il checksum dei contatori di copertura.
    for (unsigned long lind = 0; lind < seq_length; lind++)
    {
        if (seq_matches[lind] != NOT_FOUND)
        {
            checksum_matches = (checksum_matches + seq_matches[lind]) % CHECKSUM_MAX;
        }
    }

#ifdef DEBUG
    /* DEBUG: Stampa i risultati intermedi (posizioni trovate e array dei match), se in modalità DEBUG. */
    printf("-----------------\n");
    printf("Found start:");
    for (int debug_pat = 0; debug_pat < pat_number; debug_pat++)
    {
        printf(" %lu", pat_found[debug_pat]);
    }
    printf("\n");
    printf("-----------------\n");
    printf("Matches:");
    for (unsigned long lind = 0; lind < seq_length; lind++)
        printf(" %d", seq_matches[lind]);
    printf("\n");
    printf("-----------------\n");
#endif // DEBUG

    /* Liberazione delle risorse della CPU e della GPU. */
    free(sequence_h);
    // seq_matches verrà liberato alla fine del main.

    /* Liberazione della memoria allocata sulla GPU. */
    CUDA_CHECK_FUNCTION(hipFree(d_sequence));
    CUDA_CHECK_FUNCTION(hipFree(d_pat_length));
    for (int i = 0; i < pat_number; i++)
    {
        CUDA_CHECK_FUNCTION(hipFree(d_pattern_in_host[i]));
    }
    free(d_pattern_in_host);
    CUDA_CHECK_FUNCTION(hipFree(d_pattern));
    CUDA_CHECK_FUNCTION(hipFree(d_pat_found));
    CUDA_CHECK_FUNCTION(hipFree(d_seq_matches));

    /*
     *
     * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
     *
     */

    /* 8. Sincronizza la GPU per assicurarsi che tutti i calcoli siano finiti e ferma il timer. */
    CUDA_CHECK_FUNCTION(hipDeviceSynchronize());
    ttotal = cp_Wtime() - ttotal;

    /* 9. Stampa i risultati per il leaderboard. */
    printf("\n");
    /* 9.1. Tempo totale di calcolo. */
    printf("Time: %lf\n", ttotal);

    /* 9.2. Risultati: Statistiche e checksum. */
    printf("Result: %d, %lu, %lu\n\n",
           pat_matches,
           checksum_found,
           checksum_matches);

    /* 10. Liberazione finale delle risorse della CPU. */
    int i;
    for (i = 0; i < pat_number; i++)
        free(pattern[i]);
    free(pattern);
    free(pat_length);
    free(pat_found);
    free(seq_matches);

    /* 11. Fine del programma. */
    return 0;
}